#include "hip/hip_runtime.h"
/*!
 * @file cast_votes.cpp
 *
 * @author Jan Quakernack
 * @version 0.1
 */
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.hpp"


namespace igg {

__global__ void CastKernel(const long* __restrict__ kVotesXs,
                           const long* __restrict__ kVotesYs,
                           const float* __restrict__ kVotesWeights,
                           float* __restrict__ votes,
                           const int kBatchSize, const int kHeight, const int kWidth,
                           const float kThreshold) {
  const int kBatchIndex = blockIdx.y;
  const int kIndex = kBatchIndex*kHeight*kWidth+blockIdx.x*blockDim.x+threadIdx.x;

  // make sure we are inside the image
  if(kIndex>=kHeight*kWidth*kBatchSize) {return;}

  // make sure the vote is casted inside the image and weight is above threshold
  if (kVotesYs[kIndex]<0
      || kVotesYs[kIndex]>=kHeight
      || kVotesXs[kIndex]<0
      || kVotesXs[kIndex]>=kWidth
      || kVotesWeights[kIndex]<kThreshold) {return;}

  const int kVoteIndex = kBatchIndex*kHeight*kWidth+kVotesYs[kIndex]*kWidth+kVotesXs[kIndex];
  votes[kVoteIndex] += kVotesWeights[kIndex];
}


at::Tensor CastVotesCuda(torch::Tensor kVotesXs,
                         torch::Tensor kVotesYs,
                         torch::Tensor kVotesWeights,
                         const float kThreshold) {
  CHECK_DTYPE(kVotesXs, torch::kLong);
  CHECK_CONTIGUOUS(kVotesXs)
  CHECK_DIM(kVotesXs, 3);
  CHECK_DTYPE(kVotesYs, torch::kLong);
  CHECK_CONTIGUOUS(kVotesYs)
  CHECK_DIM(kVotesYs, 3);
  CHECK_DTYPE(kVotesWeights, torch::kFloat32);
  CHECK_CONTIGUOUS(kVotesWeights)
  CHECK_DIM(kVotesWeights, 3);
  CHECK_SIZE_MATCH(kVotesXs, kVotesYs, 1);
  CHECK_SIZE_MATCH(kVotesXs, kVotesYs, 2);
  CHECK_DEVICE_MATCH(kVotesXs, kVotesYs);
  CHECK_SIZE_MATCH(kVotesXs, kVotesWeights, 1);
  CHECK_SIZE_MATCH(kVotesXs, kVotesWeights, 2);
  CHECK_IS_CUDA(kVotesXs)
  CHECK_IS_CUDA(kVotesYs)
  CHECK_IS_CUDA(kVotesWeights)

  const int kBatchSize = kVotesXs.size(0);
  const int kHeight = kVotesXs.size(1);
  const int kWidth = kVotesXs.size(2);
  const int kSize = kHeight*kWidth;

  auto votes = torch::zeros_like(kVotesWeights);

  const int kNumThreads = 1024;
  // use a 2D grid of blocks, batch_index along y axis
  const dim3 kNumBlocks((kSize+kNumThreads-1)/kNumThreads, kBatchSize);

  CastKernel<<<kNumBlocks, kNumThreads>>>(kVotesXs.data<long>(), kVotesYs.data<long>(),
      kVotesWeights.data<float>(), votes.data<float>(), kBatchSize, kHeight, kWidth, kThreshold);

  return votes;
}

} // namespace igg

PYBIND11_MODULE(TORCH_EXTENSION_NAME, module) {
  module.def("cast_votes_cuda", &igg::CastVotesCuda, "Cast votes (CUDA)");
}

